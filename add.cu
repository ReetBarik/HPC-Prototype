
#include <hip/hip_runtime.h>
class Add {
	
	public:

	Add(int _a, int _b, hipStream_t _stream = 0) : 
		a(_a),
        b(_b),
        stream(_stream) {};

	int op();

	private:
	int a, b;
	hipStream_t stream;
};

__global__ void add(int a, int b, int *c) {
 		
 		*c = a + b;
}

int Add::op() {

	int c;
	int *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));
	add<<<1,1>>>(a,b,dev_c);
	hipMemcpyAsync(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost, stream);
	return c;

}

